#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <fstream>
#include <iostream>
using namespace std;


typedef struct {
  int idx;
  double *el;
} Matrix;


void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %1.9f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{

        if (argc<2) {
                cout << "Input file not specified. Please, specify it as a first argument." << endl;
                cout << "example: " << argv[0] << " transp_imgmatriz.txt" << endl;
                return -1;
        }
        ifstream file(argv[1]);
        ofstream output_S("output_S.txt");
        ofstream output_U("output_U.txt");
	ofstream output_VT("output_VT.txt");

        if (!file)
        {
                cout << "Error opening file" << endl;
                return -1;
        }

        int idx;
        file >> idx;                      //size N
        if (argc>2) cout << "N=" << idx << endl;


        hipsolverHandle_t cusolverH = NULL;
        hipblasHandle_t cublasH = NULL;
        hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
        hipError_t cudaStat1 = hipSuccess;
        hipError_t cudaStat2 = hipSuccess;
        hipError_t cudaStat3 = hipSuccess;
        hipError_t cudaStat4 = hipSuccess;
        hipError_t cudaStat5 = hipSuccess;
        hipError_t cudaStat6 = hipSuccess;
        const int m = 372; //300
        const int n = 372; //199
        const int lda = m;
   
        Matrix A, U, VT;      //host matrices
        A.el = new double[lda*n];    //.... 
        U.el = new double[lda*n];    //...
        VT.el = new double[lda*n];   //..

    //reading from file into matrices
        for (long i=0; i<(lda*n); i++){
                file >> A.el[i];
		double io = A.el[i];
                printf("%1.9f,  ",io);
	}
    printf("\n");
    
    double S[n]; // singular value
    double S_exact[n] = {7.065283497082729, 1.040081297712078};

    double *d_A = NULL;
    double *d_S = NULL;
    double *d_U = NULL;
    double *d_VT = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    double *d_rwork = NULL;
    double *d_W = NULL;  // W = S*VT

    int lwork = 0;
    int info_gpu = 0;
    const double h_one = 1;
    const double h_minus_one = -1;

 
// step 1: create cusolverDn/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double)*lda*n);
    cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(double)*n);
    cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(double)*lda*m);
    cudaStat4 = hipMalloc ((void**)&d_VT , sizeof(double)*lda*n);
    cudaStat5 = hipMalloc ((void**)&devInfo, sizeof(int));
    cudaStat6 = hipMalloc ((void**)&d_W  , sizeof(double)*lda*n);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat6);

    cudaStat1 = hipMemcpy(d_A, A.el, sizeof(double)*lda*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

// step 3: query working space of SVD
    cusolver_status = hipsolverDnDgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork );
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work , sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute SVD
    signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT
    cusolver_status = hipsolverDnDgesvd (
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        lda,
        d_S,
        d_U,
        lda,  // ldu
        d_VT,
        lda, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

 
    cudaStat1 = hipMemcpy(U.el , d_U , sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(VT.el, d_VT, sizeof(double)*lda*n, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(S , d_S , sizeof(double)*n, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    assert(0 == info_gpu);
    printf("S = (matlab base-1)\n");
    for (int k=0; k<n; k++){
                        output_S << S[k] << " ";
    }
    
    printf("=====\n");
    printf("U = (matlab base-1)\n");
    for (int k=0; k<m; k++){
                for (int j=0; j<m; j++)
                        output_U << U.el[k+j*lda] << " ";
                output_U << endl;
    }
    
    printf("=====\n");
    printf("VT = (matlab base-1)\n");
    for (int k=0; k<n; k++){
                for (int j=0; j<n; j++)
                        output_VT << VT.el[k+j*lda] << " ";
                output_VT << endl;
    }

// step 5: measure error of singular value
    double ds_sup = 0;
    for(int j = 0; j < n; j++){
        double err = fabs( S[j] - S_exact[j] );
        ds_sup = (ds_sup > err)? ds_sup : err;
    }
    printf("|S - S_exact| = %E \n", ds_sup);

// step 6: |A - U*S*VT|
    // W = S*VT
    cublas_status = hipblasDdgmm(
        cublasH,
        HIPBLAS_SIDE_LEFT,
        n,
        n,
        d_VT,
        lda,
        d_S,
         1,
        d_W,
        lda);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    // A := -U*W + A
    cudaStat1 = hipMemcpy(d_A, A.el, sizeof(double)*lda*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    cublas_status = hipblasDgemm(
        cublasH,
        HIPBLAS_OP_N, // U
        HIPBLAS_OP_N, // W
        m, // number of rows of A
        n, // number of columns of A
        n, // number of columns of U 
        &h_minus_one, /* host pointer */
        d_U, // U
        lda,
        d_W, // W
        lda,
        &h_one, /* hostpointer */
        d_A,
        lda);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    double dR_fro = 0.0;
    cublas_status = hipblasDnrm2(
        cublasH, lda*n, d_A, 1, &dR_fro);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    printf("|A - U*S*VT| = %E \n", dR_fro);

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    if (d_VT   ) hipFree(d_VT);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    if (d_W    ) hipFree(d_W);

    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return 0;
}

