/*Autores: 
*Walter Martínez Santana 
*José Carlos Castro
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multMatriz(float *da, float *db, float *dc, int num){
	float sum=0;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	while(j<num){
		while(i<num){
			for (unsigned int k = 0; k<num; k++)
				sum += da[i * num + k] * db[k * num + j];
			dc[i*num + j] = (float) sum;
			i += gridDim.y * blockDim.y;
		}
		j+=gridDim.x * blockDim.x;
		i = threadIdx.y + blockIdx.y * blockDim.y;
	}
	
}

#define n 300
#define SIZE n*n*sizeof(float)

int main(){

	int N=n;
	float *A, *B, *C;
	float *da, *db, *dc;
	int m;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	
	dim3 dimGrid(16, 16);
	dim3 dimBlock(16, 16);
	
	A=(float *)malloc(SIZE);
	B=(float *)malloc(SIZE);
	C=(float *)malloc(SIZE);
	for(m=0;m<N*N;m++){
		A[m]=(float)1;
		B[m]=(float)1;
		C[m]=(float)0;
	}
	
	hipMalloc((void**)&da, SIZE);
	hipMalloc((void**)&db, SIZE);
	hipMalloc((void**)&dc, SIZE);
	
	hipMemcpy(da,A, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(db,B, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dc,C, SIZE, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	multMatriz<<<dimGrid , dimBlock >>>(da,db,dc,N);
	//cudaThreadSynchronize();
	hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	
	hipMemcpy(C,dc, SIZE, hipMemcpyDeviceToHost);
	
	
	
	for(m=0;m<N*N;m++){
		printf("%08.0f",A[m]);
		printf("%c",( (m%N)<(N-1) ) ? '\t':'\n');
		
	}
	printf("\n\n");
	
	for(m=0;m<N*N;m++){
		printf("%08.0f",B[m]);
		printf("%c",( (m%N)<(N-1) ) ? '\t':'\n');
		
	}
	printf("\n\n");
	
	for(m=0;m<N*N;m++){
		printf("%08.0f",C[m]);
		printf("%c",( (m%N)<(N-1) ) ? '\t':'\n');
		
	} 
	printf("\n\n");
	
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Tiempo %4.6f milseg\n\n",elapsedTime);
	
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(A);
	free(B);
	free(C);
	
	return 0;
}
